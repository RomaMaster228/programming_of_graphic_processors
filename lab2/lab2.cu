#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)


// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;


__device__ double rgb_to_luma(uchar4 p) {
  return 0.299 * p.x + 0.587 * p.y + 0.114 * p.z;
}


__global__ void kernel(uchar4* out_arr, int w, int h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	for(int y = idy; y < h; y += offsety) {
	    for(int x = idx; x < w; x += offsetx) {
          double w11 = rgb_to_luma(tex2D(tex, x - 1, y - 1));
          double w12 = rgb_to_luma(tex2D(tex, x - 1, y));
          double w13 = rgb_to_luma(tex2D(tex, x - 1, y + 1));
          double w21 = rgb_to_luma(tex2D(tex, x, y - 1));
          double w22 = rgb_to_luma(tex2D(tex, x, y));
          double w23 = rgb_to_luma(tex2D(tex, x, y + 1));
          double w31 = rgb_to_luma(tex2D(tex, x + 1, y - 1));
          double w32 = rgb_to_luma(tex2D(tex, x + 1, y));
          double w33 = rgb_to_luma(tex2D(tex, x + 1, y + 1));

          double Gx = w13 + 2 * w23 + w33 - w11 - 2 * w21 - w31;
          double Gy = w31 + 2 * w32 + w33 - w11 - 2 * w12 - w13;
          int G = min(255, (int)sqrt(Gx * Gx + Gy * Gy));

          out_arr[y * w + x] = make_uchar4(G, G, G, 0);
      }
  }
}


int main() {
	int w, h;
	char input_file[100], output_file[100];
	scanf("%s", input_file);
	scanf("%s", output_file);
	FILE* fp = fopen(input_file, "rb");
	if (fp == NULL) {
	    fprintf(stderr, "Can't open file\n");
	}
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4* data = (uchar4*)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	// Подготовка данных для текстуры
	hipArray* arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));

	CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

	// Подготовка текстурной ссылки, настройка интерфейса работы с данными
	tex.addressMode[0] = hipAddressModeClamp;	// Политика обработки выхода за границы по каждому измерению
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;		// Без интерполяции при обращении по дробным координатам
	tex.normalized = false;						// Режим нормализации координат: без нормализации

	// Связываем интерфейс с данными
	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *device_out;
	CSC(hipMalloc(&device_out, sizeof(uchar4) * w * h));

	kernel<<<dim3(32, 32), dim3(32, 32)>>>(device_out, w, h);
	CSC(hipGetLastError());

	CSC(hipMemcpy(data, device_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	// Отвязываем данные от текстурной ссылки
	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(arr));
	CSC(hipFree(device_out));

	fp = fopen(output_file, "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
	return 0;
}
